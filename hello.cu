
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello()
{
  printf ("Pozdrav s GPU-a!\n");
}

// Calculates a sum of an array of integers
__global__ void sum(int *a, int *b, int *c, int n)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n)
    c[index] = a[index] + b[index];
}

// CUDA function equivalent to the Python function isSubgraph


__global__ void isSubgraph(int *result,int *graph, int *subgraph, int n, int m)
{
  int index = threadIdx.y * blockDim.x + threadIdx.x;
  if (index < n)
  {
    int i = index / m;
    int j = index % m;
    if (graph[i * m + j] != subgraph[j])
    {
      result[index] = 0;
    }
    else{
      result[index] = 1;
    }
  }
}

// Check if a subgraph is a motif by comparing it to a list of motif_candidates
__global__ void isMotif(int *subgraph, int *motif_candidates, int *result, int n)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n)
  {
    if (subgraph[0] == motif_candidates[index * 3] && subgraph[1] == motif_candidates[index * 3 + 1] && subgraph[2] == motif_candidates[index * 3 + 2])
      result[index] = 1;
  }
}